#include "hip/hip_runtime.h"


#include<stdio.h>
#include "aeslib.h"


#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                 \
        printf("code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}


__device__ aes256_context ctx;

__device__ unsigned char key[32];



__device__ void initialize(){

	for( unsigned char i = 0; i < 32; i++ ){
		key[i] = i ;
	}

}

__device__ void encrypt( unsigned char* ptr ){
	printf( "Before encry. %s\n", ptr );
	aes256_init(&ctx, key);
	aes256_encrypt_ecb(&ctx, ptr);
	printf( "Encryped on gpu %s\n", ptr );
	aes256_init(&ctx, key);
	aes256_decrypt_ecb(&ctx, ptr);
	printf( "dEncryped on gpu %s\n", ptr );
	 aes256_done(&ctx);

}


__global__ void copy( unsigned char* ptr ){
	initialize();
	encrypt( ptr );
}


int main(){

	unsigned char* array = (unsigned char* ) malloc( 7 );

	array[0] = 'u';
	array[1] = 'm';
	array[2] = 'i';
	array[3] = 't';
	array[4] = 'a';
	array[5] = 'y';
	array[6] = '\0';

	unsigned char* cc = ( unsigned char* )malloc( sizeof(array) );
	unsigned char* dev ;
    CHECK( hipMalloc( (void**)&dev, 7 ) );

	CHECK(hipMemcpy( dev, array, 7, hipMemcpyHostToDevice ));
	copy<<<1,1>>>(dev);
	CHECK(hipMemcpy( cc ,dev, 7, hipMemcpyDeviceToHost ));
	 printf("Cuda status: %s\n", hipGetErrorString( hipGetLastError() ) );



	hipDeviceReset();

	printf( "\nEncryped on gpu %s\n", cc );
	return 0;
}


#include <hip/hip_runtime.h>

#include <sys/socket.h>
#include <arpa/inet.h>
#include <unistd.h>
#include <stdio.h>
#include <netinet/if_ether.h>
#include <stdlib.h>

#define ETH_ADDR_LEN 6
#define IP_ADDR_LEN 4

const int PACKET_BUFFER_SIZE = 65536;

struct packet_info{

    unsigned char source_mac[ETH_ADDR_LEN];
    unsigned char dst_mac[ETH_ADDR_LEN];
    unsigned int source_ip;
    unsigned int dst_ip;

};



int openRawEthernetSocket(){


    int raw_socket = socket( AF_PACKET, SOCK_RAW, htons( ETH_P_ALL ) );

    if( raw_socket < 0 ){

        perror( "Could not open socket: " );
        exit( -1 );
    }


    return raw_socket;

}


__device__ void printHello(){

    printf("I am from utility function\n");
}

__device__ void resolveEthernetHeader( unsigned char* packet_buffer, struct packet_info* info )
{
	struct ethhdr *eth = ( struct ethhdr *)packet_buffer;

    info->dst_mac[0] = eth->h_dest[0];
    info->dst_mac[1] = eth->h_dest[1];
    info->dst_mac[2] = eth->h_dest[2];
    info->dst_mac[3] = eth->h_dest[3];
    info->dst_mac[4] = eth->h_dest[4];
    info->dst_mac[5] = eth->h_dest[5];

    printf(" GPU Calculated  |-Destination Address : %.2X-%.2X-%.2X-%.2X-%.2X-%.2X \n", info->dst_mac[0] , info->dst_mac[1] , info->dst_mac[2] , info->dst_mac[3] , info->dst_mac[4] , info->dst_mac[5] );
}

__global__ void processPacket( unsigned char* packet_buffer, int size, struct packet_info* info ){

    resolveEthernetHeader( packet_buffer, info );

}

int main( int argc, char* argv[] ){


	int sockaddr_size, data_size;
	struct sockaddr saddr;

	int sniffer_socket = openRawEthernetSocket();

	unsigned char* packet_buffer = new unsigned char [PACKET_BUFFER_SIZE];

    unsigned char* packet_buffer_device;

    struct packet_info* info_device;

    struct packet_info* info;

    info = ( struct packet_info* )malloc( sizeof( struct packet_info* ) );

    hipMalloc( (void**) &info_device, sizeof( struct packet_info ));

    hipMalloc( (void**) &packet_buffer_device, PACKET_BUFFER_SIZE );

  //  cudaMemset( info_device, 0, sizeof( struct packet_info ) );


	sockaddr_size = sizeof( struct sockaddr );
    dim3 block(1);
    dim3 grid(1);

	while(1){

		data_size = recvfrom( sniffer_socket, packet_buffer, PACKET_BUFFER_SIZE, 0, &saddr, (socklen_t*)&sockaddr_size );
		packet_buffer[data_size] = '\0';
        hipMemcpy( packet_buffer_device, packet_buffer ,data_size, hipMemcpyHostToDevice );
		//struct ethhdr *eth = (struct ethhdr *)packet_buffer;
		//printf(" CPU Calculated  |-Destination Address : %.2X-%.2X-%.2X-%.2X-%.2X-%.2X \n", eth->h_dest[0] , eth->h_dest[1] , eth->h_dest[2] , eth->h_dest[3] , eth->h_dest[4] , eth->h_dest[5] );
		//printf( "Packet captured\n %d \n\n", data_size );


		processPacket<<<grid, block>>>( packet_buffer_device, data_size, info_device );

        hipMemcpy( info, info_device, sizeof( struct packet_info ), hipMemcpyDeviceToHost );
        unsigned char* addr = info->dst_mac;
        printf( "READ FROM GPU -Destination Address : %.2X-%.2X-%.2X-%.2X-%.2X-%.2X \n", addr[0], addr[1], addr[2], addr[3], addr[4], addr[5] );

		//packet_buffer[data_size+1] = '\0';


	}


}

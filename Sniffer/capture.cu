#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>

#include <sys/socket.h>
#include <arpa/inet.h>
#include <unistd.h>
#include <stdio.h>
#include <netinet/if_ether.h>
#include <stdlib.h>
#include "raw_socket.h"
#include <pthread.h>

#define ETH_ADDR_LEN 6
#define IP_ADDR_LEN 4
#define THREAD_NUMBER_X 32
#define PACKET_SIZE  65536

struct packet_info{

    unsigned char source_mac[ETH_ADDR_LEN];
    unsigned char dst_mac[ETH_ADDR_LEN];
    unsigned int source_ip;
    unsigned int dst_ip;

};


static int openSnifferSocket(){

    int sniffer_socket = socket( AF_PACKET , SOCK_RAW , htons( ETH_P_ALL ) ) ;

    if( sniffer_socket == -1 ){

        perror( " Could not open sniffer socket: ");
        exit( -1 );
    }

    return sniffer_socket;
}


__device__ void resolveEthernetHeader( unsigned char* packet_buffer, struct packet_info* info )
{
   
    struct ethhdr *eth = ( struct ethhdr *)packet_buffer;

    info->dst_mac[0] = eth->h_dest[0];
    info->dst_mac[1] = eth->h_dest[1];
    info->dst_mac[2] = eth->h_dest[2];
    info->dst_mac[3] = eth->h_dest[3];
    info->dst_mac[4] = eth->h_dest[4];
    info->dst_mac[5] = eth->h_dest[5];

    printf(" GPU Resolved  |-Destination Address : %.2X-%.2X-%.2X-%.2X-%.2X-%.2X \n", info->dst_mac[0] , info->dst_mac[1] , info->dst_mac[2] , info->dst_mac[3] , info->dst_mac[4] , info->dst_mac[5] );
}

__global__ void processPacket( unsigned char* packet_buffer, int size, struct packet_info* info ){

    unsigned int thread_index = threadIdx.x;
    printf( "Thread with id %d \n", thread_index );
    packet_buffer = packet_buffer +  threadIdx.x * size;
    //resolveEthernetHeader( packet_buffer, info );

    struct ethhdr *eth = ( struct ethhdr *)packet_buffer;

       info->dst_mac[0] = eth->h_dest[0];
       info->dst_mac[1] = eth->h_dest[1];
       info->dst_mac[2] = eth->h_dest[2];
       info->dst_mac[3] = eth->h_dest[3];
       info->dst_mac[4] = eth->h_dest[4];
       info->dst_mac[5] = eth->h_dest[5];

       printf(" GPU Resolved  |-Destination Address : %.2X-%.2X-%.2X-%.2X-%.2X-%.2X \n", info->dst_mac[0] , info->dst_mac[1] , info->dst_mac[2] , info->dst_mac[3] , info->dst_mac[4] , info->dst_mac[5] );

}


static void printHeaderOnCPU( unsigned char* packet_buffer ){

	unsigned char* initial_address = packet_buffer;

	for( int i = 0; i < 31 ; i++ ){

		packet_buffer = packet_buffer + i * PACKET_SIZE;
		struct ethhdr *eth = ( struct ethhdr *)packet_buffer;


		printf( "Ethernet Header - %d -\n", i);
		printf( "   |-Destination Address : %.2X-%.2X-%.2X-%.2X-%.2X-%.2X \n", eth->h_dest[0] , eth->h_dest[1] , eth->h_dest[2] , eth->h_dest[3] , eth->h_dest[4] , eth->h_dest[5] );
		printf( "   |-Source Address      : %.2X-%.2X-%.2X-%.2X-%.2X-%.2X \n", eth->h_source[0] , eth->h_source[1] , eth->h_source[2] , eth->h_source[3] , eth->h_source[4] , eth->h_source[5] );
		packet_buffer = initial_address;

	}

	fflush( stdout );

}

int main( int argc, char* argv[] ){

    int sockaddr_size, data_size, sniffer_socket;
    int packet_counter = 0;
    int packet_index_counter = 0;
    struct sockaddr saddr; 
    unsigned char* packet_buffer;
    unsigned char* packet_buffer_device;
    struct packet_info* info_device;
    struct packet_info* info;

    packet_buffer = ( unsigned char* )malloc( sizeof( unsigned char ) * THREAD_NUMBER_X * PACKET_SIZE ) ;
    info = ( struct packet_info* )malloc( THREAD_NUMBER_X * sizeof( struct packet_info ) );
    hipMalloc( (void**) &info_device, THREAD_NUMBER_X * sizeof( struct packet_info ) );
    hipMalloc( (void**) &packet_buffer_device, THREAD_NUMBER_X * PACKET_SIZE * sizeof( unsigned char ) );


    sniffer_socket = openSnifferSocket();
    dim3 block(THREAD_NUMBER_X);
    dim3 grid(1);

    unsigned char* initial_address = packet_buffer;

    unsigned char* dummy = ( unsigned char* )malloc( sizeof( unsigned char ) * PACKET_SIZE ) ;
//
    while(1){
        data_size = recvfrom( sniffer_socket, packet_buffer, PACKET_SIZE, 0, &saddr, (socklen_t*)&sockaddr_size );


        packet_counter++;
        packet_index_counter++;
        packet_buffer = packet_buffer + PACKET_SIZE;

        if( packet_index_counter + 1 == 32 ){
        	packet_buffer = initial_address;
        	//printHeaderOnCPU( packet_buffer );
             hipMemcpy( packet_buffer_device, packet_buffer ,THREAD_NUMBER_X * PACKET_SIZE, hipMemcpyHostToDevice );
             processPacket<<<1, 32>>>( packet_buffer_device, PACKET_SIZE, info_device );
             hipMemcpy( info, info_device, sizeof( struct packet_info ), hipMemcpyDeviceToHost );
             //unsigned char* addr = info->dst_mac;
            // printf( "READ FROM GPU -Destination Address : %.2X-%.2X-%.2X-%.2X-%.2X-%.2X \n", addr[0], addr[1], addr[2], addr[3], addr[4], addr[5] );
             packet_index_counter = 0;
        }




    }


}

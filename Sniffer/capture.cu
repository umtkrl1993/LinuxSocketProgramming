#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>

#include <sys/socket.h>
#include <arpa/inet.h>
#include <unistd.h>
#include <stdio.h>
#include <netinet/if_ether.h>
#include <stdlib.h>
#include "raw_socket.h"
#include <pthread.h>

#define ETH_ADDR_LEN 6
#define IP_ADDR_LEN 4
#define THREAD_NUMBER_X 32
#define PACKET_SIZE  65536

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                 \
        printf("code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}


struct packet_info{

    unsigned char source_mac[ETH_ADDR_LEN];
    unsigned char dst_mac[ETH_ADDR_LEN];
    unsigned int source_ip;
    unsigned int dst_ip;

};


static int openSnifferSocket(){

    int sniffer_socket = socket( AF_PACKET , SOCK_RAW , htons( ETH_P_ALL ) ) ;

    if( sniffer_socket == -1 ){

        perror( " Could not open sniffer socket: ");
        exit( -1 );
    }

    return sniffer_socket;
}


__device__ void resolveEthernetHeader( unsigned char* packet_buffer, struct packet_info* info )
{
   
    struct ethhdr *eth = ( struct ethhdr *)packet_buffer;

    info->dst_mac[0] = eth->h_dest[0];
    info->dst_mac[1] = eth->h_dest[1];
    info->dst_mac[2] = eth->h_dest[2];
    info->dst_mac[3] = eth->h_dest[3];
    info->dst_mac[4] = eth->h_dest[4];
    info->dst_mac[5] = eth->h_dest[5];

    printf(" GPU Resolved  |-Destination Address : %.2X-%.2X-%.2X-%.2X-%.2X-%.2X \n", info->dst_mac[0] , info->dst_mac[1] , info->dst_mac[2] , info->dst_mac[3] , info->dst_mac[4] , info->dst_mac[5] );
}

__global__ void processPacket( unsigned char* packet_buffer, int size, struct packet_info* info, int size_info ){

    unsigned char* tmp_buffer = packet_buffer;
    struct packet_info* temp = info;

    tmp_buffer = packet_buffer +  ( threadIdx.x * size );
    temp = info + ( threadIdx.x * size_info );

    struct ethhdr *eth = ( struct ethhdr *)tmp_buffer;

    printf( "Adress of temp %p\n", temp );

    temp->source_ip = threadIdx.x;
    temp->dst_ip = threadIdx.x + 32;

    temp->dst_mac[0] = eth->h_dest[0];
    temp->dst_mac[1] = eth->h_dest[1];
    temp->dst_mac[2] = eth->h_dest[2];
    temp->dst_mac[3] = eth->h_dest[3];
    temp->dst_mac[4] = eth->h_dest[4];
    temp->dst_mac[5] = eth->h_dest[5];
    printf(" GPU Resolved with thread id %d |-Destination Address : %.2X-%.2X-%.2X-%.2X-%.2X-%.2X \n", threadIdx.x,temp->dst_mac[0] , temp->dst_mac[1] , temp->dst_mac[2] , temp->dst_mac[3] , temp->dst_mac[4] , temp->dst_mac[5] );

}


static void printHeaderOnCPU( struct packet_info* info ){

	struct packet_info * tmp = info;
	for( int i = 0; i < 32; i++ ){
		 tmp = info + ( i * sizeof( struct packet_info ));
		 printf(" CPU Resolved |-Destination Address : %.2X-%.2X-%.2X-%.2X-%.2X-%.2X \n",tmp->dst_mac[0] , tmp->dst_mac[1] , tmp->dst_mac[2] , tmp->dst_mac[3] , tmp->dst_mac[4] , tmp->dst_mac[5] );

	}

	tmp = info;
	for( int i = 0; i < 32; i++ ){
		tmp = info + ( i * sizeof (struct packet_info ) );
		printf( "resolved ids is %d and %d \n", tmp->source_ip, tmp->dst_ip );

	}
}

int main( int argc, char* argv[] ){

    int sockaddr_size, data_size, sniffer_socket;
    int packet_counter = 0;
    int packet_index_counter = 0;
    struct sockaddr saddr; 
    unsigned char* packet_buffer;
    unsigned char* packet_buffer_device;
    struct packet_info* info_device;
    struct packet_info* info;

    packet_buffer = ( unsigned char* )malloc( sizeof( unsigned char ) * THREAD_NUMBER_X * PACKET_SIZE ) ;
    info = ( struct packet_info* )malloc( THREAD_NUMBER_X * sizeof( struct packet_info ) );


    hipMalloc( (void**) &info_device, THREAD_NUMBER_X * sizeof( struct packet_info ) );
    hipMalloc( (void**) &packet_buffer_device, THREAD_NUMBER_X * PACKET_SIZE * sizeof( unsigned char ) );

    hipMemset( info_device, THREAD_NUMBER_X* sizeof( struct packet_info ), 0 );


    sniffer_socket = openSnifferSocket();
    dim3 block(THREAD_NUMBER_X);
    dim3 grid(1);

    unsigned char* initial_address = packet_buffer;

    unsigned char* dummy = ( unsigned char* )malloc( sizeof( unsigned char ) * PACKET_SIZE ) ;

    int size_info = sizeof( struct packet_info );

//
    while(1){
        data_size = recvfrom( sniffer_socket, packet_buffer, PACKET_SIZE, 0, &saddr, (socklen_t*)&sockaddr_size );


        packet_counter++;
        packet_index_counter++;
        packet_buffer = packet_buffer + PACKET_SIZE;

        if( packet_index_counter + 1 == 32 ){
        	packet_buffer = initial_address;
        	//printHeaderOnCPU( packet_buffer );
             hipMemcpy( packet_buffer_device, packet_buffer ,THREAD_NUMBER_X * PACKET_SIZE, hipMemcpyHostToDevice );
             processPacket<<<grid, 16>>>( packet_buffer_device, PACKET_SIZE, info_device, sizeof( struct packet_info ) );

             CHECK(hipMemcpy( info , info_device,THREAD_NUMBER_X * sizeof(struct packet_info), hipMemcpyDeviceToHost ));
             CHECK(hipGetLastError());
             hipFree( info_device );
             hipFree( packet_buffer_device );
             hipDeviceReset();
             printHeaderOnCPU( info );
             //unsigned char* addr = info->dst_mac;
            // printf( "READ FROM GPU -Destination Address : %.2X-%.2X-%.2X-%.2X-%.2X-%.2X \n", addr[0], addr[1], addr[2], addr[3], addr[4], addr[5] );
             packet_index_counter = 0;

             exit(0);
        }




    }


}
